#include <hip/hip_runtime.h>
#include <iostream>
#include <chrono>
#include <thread>

// CUDA kernel
__global__ void vectorAdd(const float* a, const float* b, float* c, int N) {
    int i = threadIdx.x + blockIdx.x * blockDim.x;
    if (i < N) {
        c[i] = a[i] + b[i];
    }
}

int main() {
    const int N = 1024;
    const int size = N * sizeof(float);

    // Host memory allocation
    float *h_a, *h_b, *h_c;
    h_a = (float*)malloc(size);
    h_b = (float*)malloc(size);
    h_c = (float*)malloc(size);

    // Initialize host arrays
    for (int i = 0; i < N; ++i) {
        h_a[i] = i * 1.0f;
        h_b[i] = i * 2.0f;
    }

    // Device memory allocation
    float *d_a, *d_b, *d_c;
    hipMalloc((void**)&d_a, size);
    hipMalloc((void**)&d_b, size);
    hipMalloc((void**)&d_c, size);

    // Copy data from host to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);

    // Define thread hierarchy
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    while (true) {
        // Launch kernel
        vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, N);

        // Check for kernel errors
        hipError_t kernel_err = hipGetLastError();
        if (kernel_err != hipSuccess) {
            std::cerr << "CUDA kernel launch error: " << hipGetErrorString(kernel_err) << std::endl;
            break;
        }

        // Copy results back to host
        hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);

        // Verify and output results
        bool success = true;
        std::cout << "Results (First 10 Elements):" << std::endl;
        for (int i = 0; i < 10; ++i) { // Print first 10 results
            std::cout << "c[" << i << "] = " << h_c[i] << " (Expected: " << h_a[i] + h_b[i] << ")" << std::endl;
            if (h_c[i] != h_a[i] + h_b[i]) {
                success = false;
            }
        }

        if (!success) {
            std::cerr << "Results verification failed!" << std::endl;
            break;
        } else {
            std::cout << "Results verification succeeded!" << std::endl;
        }

        // Memory usage report
        size_t freeMem, totalMem;
        hipMemGetInfo(&freeMem, &totalMem);
        std::cout << "Memory Report:" << std::endl;
        std::cout << "  Free memory: " << freeMem / (1024.0 * 1024.0) << " MB" << std::endl;
        std::cout << "  Total memory: " << totalMem / (1024.0 * 1024.0) << " MB" << std::endl;
        std::cout << "  Memory used: " << (totalMem - freeMem) / (1024.0 * 1024.0) << " MB" << std::endl;

        // Wait for 90 seconds before the next iteration
        std::cout << "Waiting for 90 seconds..." << std::endl;
        std::this_thread::sleep_for(std::chrono::seconds(90));
    }

    // Free memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);

    std::cout << "Exiting program..." << std::endl;
    return 0;
}
